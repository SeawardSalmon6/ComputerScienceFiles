#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *result) {
  const int idx = threadIdx.x;
  result[idx] = a[idx] + b[idx];
}

int main() {
  int *a, *b, *result;
  int *dev_a, *dev_b, *dev_result;
  int size = THREADS_PER_BLOCK * sizeof(int);

  // Aloca os arrays no Device
  hipMalloc((void **)&dev_a, size);
  hipMalloc((void **)&dev_b, size);
  hipMalloc((void **)&dev_result, size);

  a = (int *)malloc(size);
  b = (int *)malloc(size);
  result = (int *)malloc(size);

  srand(time(NULL));
  for (int i = 0; i < THREADS_PER_BLOCK; i++) {
    a[i] = i + 1;
    b[i] = i + 1;
  }

  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

  add << <1, THREADS_PER_BLOCK >> > (dev_a, dev_b, dev_result);

  hipMemcpy(result, dev_result, size, hipMemcpyDeviceToHost);

  printf("[");
  for (int i = 0; i < THREADS_PER_BLOCK; i++) {
    printf("%d, ", a[i]);
  }
  printf("]\n");

  printf("[");
  for (int i = 0; i < THREADS_PER_BLOCK; i++) {
    printf("%d, ", b[i]);
  }
  printf("]\n");

  printf("[");
  for (int i = 0; i < THREADS_PER_BLOCK; i++) {
    printf("%d, ", result[i]);
  }
  printf("]\n");

  free(a);
  free(b);
  free(result);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_result);

  return 0;
}
