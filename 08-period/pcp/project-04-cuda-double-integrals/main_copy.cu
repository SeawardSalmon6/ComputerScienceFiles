#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define LOGGER_FILENAME "log.csv"

#define LOWER_BOUND 0
#define UPPER_BOUND 1.5

#define ROOT_PROCESS 0
#define THREADS_PER_BLOCK 512

#define BLOCKS 10
// #define BLOCKS 100
// #define BLOCKS 1000

#define TEST_NUM 1
const long X_POINTS = (long)1e3; // 10^3
const long Y_POINTS = (long)1e3; // 10^3

// #define TEST_NUM 2
// const long X_POINTS = (long)1e3; // 10^3
// const long Y_POINTS = (long)1e4; // 10^4

// #define TEST_NUM 3
// const long X_POINTS = (long)1e3; // 10^3
// const long Y_POINTS = (long)1e5; // 10^5

/* ==================================== */

// #define TEST_NUM 4
// const long X_POINTS = (long)1e4; // 10^4
// const long Y_POINTS = (long)1e3; // 10^3

// #define TEST_NUM 5
// const long X_POINTS = (long)1e4; // 10^4
// const long Y_POINTS = (long)1e4; // 10^4

// #define TEST_NUM 6
// const long X_POINTS = (long)1e4; // 10^4
// const long Y_POINTS = (long)1e5; // 10^5

/* ==================================== */

// #define TEST_NUM 7
// const long X_POINTS = (long)1e5; // 10^5
// const long Y_POINTS = (long)1e3; // 10^3

// #define TEST_NUM 8
// const long X_POINTS = (long)1e5; // 10^5
// const long Y_POINTS = (long)1e4; // 10^4

// #define TEST_NUM 9
// const long X_POINTS = (long)1e5; // 10^5
// const long Y_POINTS = (long)1e5; // 10^5

__device__ double f(const double x, const double y) {
  // f(x, y) = sin(x² + y²)
  return sin(x * x + y * y);
}

__global__ void double_trapz(double *global_result) {
  const long X_INTERVALS = X_POINTS - 1;
  const long Y_INTERVALS = Y_POINTS - 1;

  const double HX = (UPPER_BOUND - LOWER_BOUND) / X_INTERVALS;
  const double HX_HALF = HX / 2.0;
  const double HY = (UPPER_BOUND - LOWER_BOUND) / Y_INTERVALS;
  const double HY_HALF = HY / 2.0;

  const long idx = blockIdx.x * blockDim.x + threadIdx.x;
  long intervals_per_thread = (X_INTERVALS * Y_INTERVALS) / (BLOCKS * THREADS_PER_BLOCK);
  long outer_start = idx * intervals_per_thread;
  long outer_end = (idx + 1) * intervals_per_thread;

  double x, inner_result, fxy;
  double local_result = 0.0;

  for (long i = outer_start; i < outer_end; i++) {
    x = LOWER_BOUND + i * HX;
    inner_result = 0.0;

    for (long j = 0; j < Y_POINTS; j++) {
      fxy = f(x, LOWER_BOUND + j * HY);
      inner_result += (j == 0 || j == Y_INTERVALS) ? fxy : 2 * fxy;
    }

    inner_result *= HY_HALF;
    local_result += (i == 0 || i == X_INTERVALS) ? inner_result : 2 * inner_result;
  }

  *global_result += local_result * HX_HALF;
}

int main(int argc, char *argv[]) {
  double result, time;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  double_trapz<<<BLOCKS, THREADS_PER_BLOCK>>>(&result);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  FILE *logger_fp = fopen(LOGGER_FILENAME, "a+");

  fprintf(logger_fp, "\"%d\";", TEST_NUM);    // Test ID
  fprintf(logger_fp, "\"%d\";", BLOCKS);      // Blocks length
  fprintf(logger_fp, "\"%ld\";", X_POINTS);   // X points
  fprintf(logger_fp, "\"%ld\";", Y_POINTS);   // Y points
  fprintf(logger_fp, "\"%.16lf\";", result);  // Result
  fprintf(logger_fp, "\"%.16lf\";\n", time);  // Time elapsed

  fclose(logger_fp);

  return 0;
}
